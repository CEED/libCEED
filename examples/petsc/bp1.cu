#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory. LLNL-CODE-734707.
// All Rights reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

// *****************************************************************************
extern "C" __global__ void Setup(void *ctx, CeedInt Q,
                                 Fields_Cuda fields) {
  CeedScalar *rho = fields.outputs[0], *true_soln = fields.outputs[1], *rhs = fields.output[2];
  const CeedScalar (*x)[Q] = (const CeedScalar (*)[Q])fields.inputs[0];
  const CeedScalar (*J)[3][Q] = (const CeedScalar (*)[3][Q])fields.inputs[1];
  const CeedScalar *w = in[2];
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < Q;
       i += blockDim.x * gridDim.x) {
    CeedScalar det = (+ J[0][0][i] * (J[1][1][i]*J[2][2][i] - J[1][2][i]*J[2][1][i])
                      - J[0][1][i] * (J[1][0][i]*J[2][2][i] - J[1][2][i]*J[2][0][i])
                      + J[0][2][i] * (J[1][0][i]*J[2][1][i] - J[1][1][i]*J[2][0][i]));
    rho[i] = det * w[i];
    true_soln[i] = sqrt(x[0][i]*x[0][i] + x[1][i]*x[1][i] + x[2][i]*x[2][i]);
    rhs[i] = rho[i] * true_soln[i];
  }
}

extern "C" __global__ void Mass(void *ctx, CeedInt Q,
                Fields_Cuda fields) {
  const CeedScalar *u = fields.inputs[0], *rho = fields.inputs[1];
  CeedScalar *v = fields.outputs[0];
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < Q;
       i += blockDim.x * gridDim.x) {
    v[i] = rho[i] * u[i];
  }
}

extern "C" __global__ void Error(void *ctx, CeedInt Q,
                                 Fields_Cuda fields) {
  const CeedScalar *u = fields.inputs[0], *target = fields.inputs[1];
  CeedScalar *err = fields.outputs[0];
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < Q;
       i += blockDim.x * gridDim.x) {
    err[i] = u[i] - target[i];
  }
}
