#include "hip/hip_runtime.h"
// *****************************************************************************
extern "C" __global__ void setup(void *ctx, CeedInt Q, const CeedScalar *const *in,
                      CeedScalar *const *out) {
  const CeedScalar *w = in[0];
  CeedScalar *qdata = out[0];
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
    i < Q;
    i += blockDim.x * gridDim.x)
  {
    qdata[i] = w[i];
  }
}

// *****************************************************************************
extern "C" __global__ void mass(void *ctx, CeedInt Q, const CeedScalar *const *in,
                     CeedScalar *const *out) {
  const CeedScalar *qdata = in[0], *u = in[1];
  CeedScalar *v = out[0];
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
    i < Q;
    i += blockDim.x * gridDim.x)
  {
    v[i] = qdata[i] * u[i];
  }
}

