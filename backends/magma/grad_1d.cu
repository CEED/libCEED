#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory. LLNL-CODE-734707.
// All Rights reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

#include <ceed.h>
#include <hip/hip_runtime.h>    // for CUDA_VERSION
#include <magma_v2.h>
#include "magma_common_device.cuh"
#include "grad_device.cuh"

//////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ CeedScalar shared_data[];
template<typename T, int DIM, int NCOMP, int P, int Q>
static __global__ void
magma_grad_1d_kernel(  
    const T *dTgrad, magma_trans_t transT,
    const T *dU, const int estrdU, const int cstrdU, 
          T *dV, const int estrdV, const int cstrdV, const int nelem)
{
    const int tx      = threadIdx.x;
    const int ty      = threadIdx.y;
    const int elem_id = (blockIdx.x * blockDim.y) + ty;

    if (elem_id >= nelem) return;

    T* sU[NCOMP];
    T* sV[NCOMP];

    // shift global memory pointers by elem stride
    dU += elem_id * estrdU;
    dV += elem_id * estrdV;

    // assign shared memory pointers
    T* sT = (T*)(shared_data);
    T* sW = sT + P*Q;
    sU[0] = sW + ty * NCOMP * (P + Q);
    sV[0] = sU[0] + (NCOMP * 1 * P);
    for(int icomp = 1; icomp < NCOMP; icomp++) {
        sU[icomp] = sU[icomp-1] + (1 * P);
        sV[icomp] = sV[icomp-1] + (1 * Q);
    }

    // read T
    if (ty == 0) {
        dread_T_gm2sm<P, Q>(tx, transT, dTgrad, sT);
    }

    // read U
    read_1d<T, P, NCOMP>(dU, cstrdU, sU, tx);

    // read V if transT is magmaTrans
    if (transT == MagmaTrans) {
        read_1d<T, Q, NCOMP>(dV, cstrdV, sV, tx);
    }

    __syncthreads();
    magma_grad_1d_device<T, DIM, NCOMP, P, Q>(sT, transT, sU, sV, tx);
    __syncthreads();

    // write V
    write_1d<T, Q, NCOMP>(sV, dV, cstrdV, tx);
}

//////////////////////////////////////////////////////////////////////////////////////////
template<typename T, int NCOMP, int P, int Q>
static magma_int_t 
magma_grad_1d_kernel_driver(  
                const T *dTgrad, magma_trans_t transT,
                const T *dU, magma_int_t estrdU, magma_int_t cstrdU, 
                      T *dV, magma_int_t estrdV, magma_int_t cstrdV, 
                magma_int_t nelem, magma_int_t maxthreads, magma_queue_t queue)
{
    magma_device_t device;
    magma_getdevice( &device );
    magma_int_t shmem_max, nthreads_max;

    magma_int_t nthreads = max(P, Q); 
    magma_int_t ntcol = (maxthreads < nthreads) ? 1 : (maxthreads / nthreads);
    magma_int_t shmem  = 0;
    shmem += sizeof(T) * ntcol * (NCOMP * (1*P + 1*Q)); 
    shmem += sizeof(T) * (P*Q);    

    hipDeviceGetAttribute (&nthreads_max, hipDeviceAttributeMaxThreadsPerBlock, device);
    #if CUDA_VERSION >= 9000
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeSharedMemPerBlockOptin, device);
    if (shmem <= shmem_max) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(magma_grad_1d_kernel<T), 1, NCOMP, P, Q>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
    }
    #else
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    #endif    // CUDA_VERSION >= 9000
 
    if ( (nthreads*ntcol) > nthreads_max || shmem > shmem_max ) {
        return 1;    // launch failed
    }
    else {
        magma_int_t nblocks = (nelem + ntcol-1) / ntcol;
        dim3 threads(nthreads, ntcol, 1);
        dim3 grid(nblocks, 1, 1);
        magma_grad_1d_kernel<T, 1, NCOMP, P, Q><<<grid, threads, shmem, magma_queue_get_cuda_stream(queue)>>>
        (dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem);
        return (hipPeekAtLastError() == hipSuccess) ? 0 : 1;
    }
}

//////////////////////////////////////////////////////////////////////////////////////////
template<int P, int Q>
static magma_int_t 
magma_grad_1d_ncomp(
                magma_int_t ncomp,
                const CeedScalar *dTgrad, magma_trans_t transT,
                const CeedScalar *dU, magma_int_t estrdU, magma_int_t cstrdU, 
                      CeedScalar *dV, magma_int_t estrdV, magma_int_t cstrdV, 
                magma_int_t nelem, magma_int_t maxthreads, magma_queue_t queue)
{
    magma_int_t launch_failed = 0;
    switch (ncomp) {
        case 1: 
          launch_failed = magma_grad_1d_kernel_driver<CeedScalar,1,P,Q>
          (dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case 2: 
          launch_failed = magma_grad_1d_kernel_driver<CeedScalar,2,P,Q>
          (dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case 3: 
          launch_failed = magma_grad_1d_kernel_driver<CeedScalar,3,P,Q>
          (dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        default: launch_failed = 1;
    }
    return launch_failed;
}

//////////////////////////////////////////////////////////////////////////////////////////
template<int P>
static magma_int_t 
magma_grad_1d_ncomp_q(
                magma_int_t Q, magma_int_t ncomp,
                const CeedScalar *dTgrad, magma_trans_t transT,
                const CeedScalar *dU, magma_int_t estrdU, magma_int_t cstrdU, 
                      CeedScalar *dV, magma_int_t estrdV, magma_int_t cstrdV, 
                magma_int_t nelem, magma_int_t maxthreads, magma_queue_t queue)
{
    magma_int_t launch_failed = 0;
    switch (Q) {
        case  1: 
          launch_failed = magma_grad_1d_ncomp<P, 1>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  2: 
          launch_failed = magma_grad_1d_ncomp<P, 2>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  3: 
          launch_failed = magma_grad_1d_ncomp<P, 3>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  4: 
          launch_failed = magma_grad_1d_ncomp<P, 4>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  5: 
          launch_failed = magma_grad_1d_ncomp<P, 5>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  6: 
          launch_failed = magma_grad_1d_ncomp<P, 6>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  7: 
          launch_failed = magma_grad_1d_ncomp<P, 7>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  8: 
          launch_failed = magma_grad_1d_ncomp<P, 8>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  9: 
          launch_failed = magma_grad_1d_ncomp<P, 9>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case 10: 
          launch_failed = magma_grad_1d_ncomp<P,10>
          (ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        default: launch_failed = 1;
    }
    return launch_failed;
}


//////////////////////////////////////////////////////////////////////////////////////////
static magma_int_t 
magma_grad_1d_ncomp_q_p(
                magma_int_t P, magma_int_t Q, magma_int_t ncomp,
                const CeedScalar *dTgrad, magma_trans_t transT,
                const CeedScalar *dU, magma_int_t estrdU, magma_int_t cstrdU, 
                      CeedScalar *dV, magma_int_t estrdV, magma_int_t cstrdV, 
                magma_int_t nelem, magma_int_t maxthreads, magma_queue_t queue)
{
    magma_int_t launch_failed = 0;
    switch (P) {
        case  1: 
          launch_failed = magma_grad_1d_ncomp_q< 1>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  2: 
          launch_failed = magma_grad_1d_ncomp_q< 2>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  3: 
          launch_failed = magma_grad_1d_ncomp_q< 3>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  4: 
          launch_failed = magma_grad_1d_ncomp_q< 4>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  5: 
          launch_failed = magma_grad_1d_ncomp_q< 5>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  6: 
          launch_failed = magma_grad_1d_ncomp_q< 6>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  7: 
          launch_failed = magma_grad_1d_ncomp_q< 7>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  8: 
          launch_failed = magma_grad_1d_ncomp_q< 8>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case  9: 
          launch_failed = magma_grad_1d_ncomp_q< 9>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        case 10: 
          launch_failed = magma_grad_1d_ncomp_q<10>
          (Q, ncomp, dTgrad, transT, dU, estrdU, cstrdU, dV, estrdV, cstrdV, nelem, maxthreads, queue); 
          break;
        default: launch_failed = 1;
    }
    return launch_failed;
}



//////////////////////////////////////////////////////////////////////////////////////////
extern "C" magma_int_t 
magma_grad_1d( 
    magma_int_t P, magma_int_t Q, magma_int_t ncomp,  
    const CeedScalar *dTinterp, const CeedScalar *dTgrad, CeedTransposeMode tmode,
    const CeedScalar *dU, magma_int_t estrdU, magma_int_t cstrdU, 
          CeedScalar *dV, magma_int_t estrdV, magma_int_t cstrdV, 
    magma_int_t nelem, magma_int_t maxthreads, magma_queue_t queue)
{    
    magma_int_t launch_failed = 0;
    magma_trans_t transT = (tmode == CEED_NOTRANSPOSE) ? MagmaNoTrans : MagmaTrans;
    launch_failed = magma_grad_1d_ncomp_q_p(
                        P, Q, ncomp, 
                        dTgrad, transT, 
                        dU, estrdU, cstrdU, 
                        dV, estrdV, cstrdV, 
                        nelem, maxthreads, queue);

    return launch_failed;
}
