#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory. LLNL-CODE-734707.
// All Rights reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

#include <ceed.h>
#include <hip/hip_runtime.h>    // for CUDA_VERSION
#include <magma_v2.h>
#include "magma_tc_device.cuh"
#include "interp_device.cuh"

//////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ CeedScalar shared_data[];
template<typename T, int DIM, int NCOMP, int P, int Q>
static __global__ void
magma_interp_1d_kernel(  
    const T *dT, magma_trans_t transT,
    const T *dU, const int u_elstride, const int u_compstride, 
          T *dV, const int v_elstride, const int v_compstride)
{
    const int elem_id = blockIdx.x;
    const int tx      = threadIdx.x;
    T* sU[NCOMP];
    T* sV[NCOMP];

    // shift global memory pointers by elem stride
    dU += elem_id * u_elstride;
    dV += elem_id * v_elstride;

    // assign shared memory pointers
    T* sT = (T*)(shared_data);
    sU[0] = sT + P*Q;
    sV[0] = sU[0] + (NCOMP * 1 * P);
    for(int icomp = 1; icomp < NCOMP; icomp++) {
        sU[icomp] = sU[icomp-1] + (1 * P);
        sV[icomp] = sV[icomp-1] + (1 * Q);
    }

    // read T
    if( transT == MagmaNoTrans ) {
        // T is P x Q
        if(tx < P) {
            for(int j = 0; j < Q; j++) {
                sT[j * P + tx] = dT[j	 * P + tx];
            }
        }
    }
    else {
        // T is Q x P -- transpose it in shared memory
        if(tx < Q) {
            #pragma unroll
            for(int i = 0; i < P; i++) {
                sT[tx * P + i] = dT[i * Q + tx];
            }
        }
    }

    // read U
    if(tx < P) {
        for(int icomp = 0; icomp < NCOMP; icomp++) {
            sU[icomp][tx] = dU[icomp * u_compstride + tx];
        }
    }

    // read V if transT is magmaTrans
    if(transT == MagmaTrans) {
        if(tx < Q) {
            for(int icomp = 0; icomp < NCOMP; icomp++) {
                sV[icomp][tx] = dV[icomp * v_compstride + tx];
            }
        }
    }
    __syncthreads();

    magma_interp_1d_device<T, DIM, NCOMP, P, Q>(sT, transT, sU, sV, tx);
    __syncthreads();

    // write V
    if(tx < Q) {
        for(int icomp = 0; icomp < NCOMP; icomp++) {
            dV[icomp * v_compstride + tx] = sV[icomp][tx];
        }
    }
}

//////////////////////////////////////////////////////////////////////////////////////////
template<typename T, int NCOMP, int P, int Q>
static magma_int_t 
magma_interp_1d_kernel_driver(  
                const T *dT, magma_trans_t transT,
                const T *dU, magma_int_t u_elstride, magma_int_t u_compstride, 
                      T *dV, magma_int_t v_elstride, magma_int_t v_compstride, 
                magma_int_t nelem, magma_queue_t queue)
{
    magma_device_t device;
    magma_getdevice( &device );
    magma_int_t shmem_max, nthreads_max;

    magma_int_t shmem  = 0;
    shmem += NCOMP * (1*P + 1*Q) * sizeof(T); 
    shmem += P*Q*sizeof(T);    
    magma_int_t nthreads = max(P, Q); 

    hipDeviceGetAttribute (&nthreads_max, hipDeviceAttributeMaxThreadsPerBlock, device);
    #if CUDA_VERSION >= 9000
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeSharedMemPerBlockOptin, device);
    if(shmem <= shmem_max) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(magma_interp_1d_kernel<T), 1, NCOMP, P, Q>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
    }
    #else
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    #endif    // CUDA_VERSION >= 9000
 
    if( nthreads > nthreads_max || shmem > shmem_max ) {
        return 1;    // launch failed
    }
    else { 
        dim3 threads(nthreads, 1, 1);
        dim3 grid(nelem, 1, 1);
        magma_interp_1d_kernel<T, 1, NCOMP, P, Q><<<grid, threads, shmem, magma_queue_get_cuda_stream(queue)>>>
        (dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride);
        return (hipPeekAtLastError() == hipSuccess) ? 0 : 1;
    }
}

//////////////////////////////////////////////////////////////////////////////////////////
template<int P, int Q>
static magma_int_t 
magma_interp_1d_ncomp(
                magma_int_t ncomp,
                const CeedScalar *dT, magma_trans_t transT,
                const CeedScalar *dU, magma_int_t u_elstride, magma_int_t u_compstride, 
                      CeedScalar *dV, magma_int_t v_elstride, magma_int_t v_compstride, 
                magma_int_t nelem, magma_queue_t queue)
{
    magma_int_t launch_failed = 0;
    switch(ncomp) {
        case 1: launch_failed = magma_interp_1d_kernel_driver<CeedScalar,1,P,Q>(dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 2: launch_failed = magma_interp_1d_kernel_driver<CeedScalar,2,P,Q>(dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 3: launch_failed = magma_interp_1d_kernel_driver<CeedScalar,3,P,Q>(dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        default: launch_failed = 1;
    }
    return launch_failed;
}

//////////////////////////////////////////////////////////////////////////////////////////
template<int P>
static magma_int_t 
magma_interp_1d_ncomp_q(
                magma_int_t Q, magma_int_t ncomp,
                const CeedScalar *dT, magma_trans_t transT,
                const CeedScalar *dU, magma_int_t u_elstride, magma_int_t u_compstride, 
                      CeedScalar *dV, magma_int_t v_elstride, magma_int_t v_compstride, 
                magma_int_t nelem, magma_queue_t queue)
{
    magma_int_t launch_failed = 0;
    switch(Q) {
        case  1: launch_failed = magma_interp_1d_ncomp<P, 1>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  2: launch_failed = magma_interp_1d_ncomp<P, 2>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  3: launch_failed = magma_interp_1d_ncomp<P, 3>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  4: launch_failed = magma_interp_1d_ncomp<P, 4>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  5: launch_failed = magma_interp_1d_ncomp<P, 5>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  6: launch_failed = magma_interp_1d_ncomp<P, 6>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  7: launch_failed = magma_interp_1d_ncomp<P, 7>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  8: launch_failed = magma_interp_1d_ncomp<P, 8>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  9: launch_failed = magma_interp_1d_ncomp<P, 9>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 10: launch_failed = magma_interp_1d_ncomp<P,10>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 11: launch_failed = magma_interp_1d_ncomp<P,11>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 12: launch_failed = magma_interp_1d_ncomp<P,12>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 13: launch_failed = magma_interp_1d_ncomp<P,13>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 14: launch_failed = magma_interp_1d_ncomp<P,14>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 15: launch_failed = magma_interp_1d_ncomp<P,15>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 16: launch_failed = magma_interp_1d_ncomp<P,16>(ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        default: launch_failed = 1;
    }
    return launch_failed;
}


//////////////////////////////////////////////////////////////////////////////////////////
static magma_int_t 
magma_interp_1d_ncomp_q_p(
                magma_int_t P, magma_int_t Q, magma_int_t ncomp,
                const CeedScalar *dT, magma_trans_t transT,
                const CeedScalar *dU, magma_int_t u_elstride, magma_int_t u_compstride, 
                      CeedScalar *dV, magma_int_t v_elstride, magma_int_t v_compstride, 
                magma_int_t nelem, magma_queue_t queue)
{
    magma_int_t launch_failed = 0;
    switch(P) {
        case  1: launch_failed = magma_interp_1d_ncomp_q< 1>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  2: launch_failed = magma_interp_1d_ncomp_q< 2>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  3: launch_failed = magma_interp_1d_ncomp_q< 3>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  4: launch_failed = magma_interp_1d_ncomp_q< 4>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  5: launch_failed = magma_interp_1d_ncomp_q< 5>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  6: launch_failed = magma_interp_1d_ncomp_q< 6>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  7: launch_failed = magma_interp_1d_ncomp_q< 7>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  8: launch_failed = magma_interp_1d_ncomp_q< 8>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case  9: launch_failed = magma_interp_1d_ncomp_q< 9>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 10: launch_failed = magma_interp_1d_ncomp_q<10>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 11: launch_failed = magma_interp_1d_ncomp_q<11>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 12: launch_failed = magma_interp_1d_ncomp_q<12>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 13: launch_failed = magma_interp_1d_ncomp_q<13>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 14: launch_failed = magma_interp_1d_ncomp_q<14>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 15: launch_failed = magma_interp_1d_ncomp_q<15>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        case 16: launch_failed = magma_interp_1d_ncomp_q<16>(Q, ncomp, dT, transT, dU, u_elstride, u_compstride, dV, v_elstride, v_compstride, nelem, queue); break;
        default: launch_failed = 1;
    }
    return launch_failed;
}



//////////////////////////////////////////////////////////////////////////////////////////
extern "C" magma_int_t 
magma_interp_1d( 
    magma_int_t P, magma_int_t Q, magma_int_t ncomp,  
    const CeedScalar *dT, CeedTransposeMode tmode,
    const CeedScalar *dU, magma_int_t u_elstride, magma_int_t u_compstride, 
          CeedScalar *dV, magma_int_t v_elstride, magma_int_t v_compstride, 
    magma_int_t nelem, magma_queue_t queue)
{    
    magma_int_t launch_failed = 0;
    magma_trans_t transT = (tmode == CEED_NOTRANSPOSE) ? MagmaNoTrans : MagmaTrans;
    launch_failed = magma_interp_1d_ncomp_q_p(
                        P, Q, ncomp, 
                        dT, transT, 
                        dU, u_elstride, u_compstride, 
                        dV, v_elstride, v_compstride, 
                        nelem, queue);

    return launch_failed;
}
