#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory. LLNL-CODE-734707.
// All Rights reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

#include <ceed.h>
#include <magma.h>

//////////////////////////////////////////////////////////////////////////////////////////

// dv(i, c, e) = du( ind(i, e), c)  
static __global__ void 
magma_readDofs_kernel(const int NCOMP, const int nnodes, const int nelem,
                      int *indices, 
                      const double *du, double *dv)
{
  const int  pid = threadIdx.x;
  const int elem = blockIdx.x;
 
  for (CeedInt i = pid; i < nnodes; i += blockDim.x) {
        const CeedInt ind = indices ? indices[i + elem * nnodes] : i + elem * nnodes;
        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            // dv[i+comp*nnodes+elem*NCOMP*nnodes] = du[ind + nnodes * comp];
            dv[i+elem*nnodes+comp*nnodes*nelem] = du[ind + nnodes * comp];
        }
  }
}

// dv(i, c, e) = du( c, ind(i,e))  
static __global__ void
magma_readDofsTranspose_kernel(const int NCOMP, const int nnodes, const int nelem,
                               int *indices,
                               const double *du, double *dv)
{
    const int  pid = threadIdx.x;
    const int elem = blockIdx.x;

    CeedInt   cb = pid%NCOMP;
    CeedInt   tb = blockDim.x;
    __shared__ CeedScalar dofs[tb][NCOMP];
    __shared__ const CeedInt  ind[nnodes];
    for (CeedInt i = pid; i < nnodes; i += tb) {
        ind[i] = indices ? indices[i + elem * nnodes] : i + elem * nnodes;

        __syncthreads();

        for (CeedInt j = i/NCOMP; j<min(tb, nnodes); j+=NCOMP)
            dofs[j][cb] = du[cb + ind[j] * NCOMP];

        __syncthreads();

        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            // dv[i+comp*nnodes+elem*NCOMP*nnodes] = dofs[i][comp];
            dv[i+elem*nnodes+comp*nnodes*nelem] = dofs[i][comp];
        }
    }
}

// dv( ind(i, e), c) = du(i, c, e) 
static __global__ void 
magma_writeDofs_kernel(const int NCOMP, const int nnodes, const int nelem,
                      int *indices, 
                      const double *du, double *dv)
{
    const int  pid = threadIdx.x;
    const int elem = blockIdx.x;

    for (CeedInt i = pid; i < nnodes; i += blockDim.x) {
        const CeedInt ind = indices ? indices[i + elem * nnodes] : i + elem * nnodes;
        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            // magmablas_datomic_add(&dv[ind + nnodes * comp], 
            //                       du[i+comp*nnodes+elem*NCOMP*nnodes]);
            magmablas_datomic_add(&dv[ind + nnodes * comp],
                                  du[i+elem*nnodes+comp*nnodes*nelem]);
        }
    }
}

// dv( c, ind(i,e)) = du(i, c, e)
static __global__ void
magma_writeDofsTranspose_kernel(const int NCOMP, const int nnodes, const int nelem,
                               int *indices,
                               const double *du, double *dv)
{
    const int  pid = threadIdx.x;
    const int elem = blockIdx.x;

    CeedInt   cb = pid%NCOMP;
    CeedInt   tb = blockDim.x;
    __shared__ CeedScalar dofs[tb][NCOMP];
    __shared__ const CeedInt  ind[nnodes];
    for (CeedInt i = pid; i < nnodes; i += tb) {
        ind[i] = indices ? indices[i + elem * nnodes] : i + elem * nnodes;

        __syncthreads();
        
        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            dofs[i][comp] = du[i+comp*nnodes+elem*NCOMP*nnodes];
            dofs[i][comp] = du[i+elem*nnodes+comp*nnodes*nelem];
        }

        __syncthreads();

        for (CeedInt j = i/NCOMP; j<min(tb, nnodes); j+=NCOMP)
            magmablas_datomic_add(&dv[cb + ind[j] * NCOMP], dofs[j][cb]);
    }
}


//////////////////////////////////////////////////////////////////////////////////////////

// ReadDofs to device memory in tensor dv of size nnodes x NCOMP x nelem
// dv(i, c, e) = du( ind(i, e), c)    
extern "C" void
magma_readDofs(const magma_int_t NCOMP, 
               const magma_int_t nnodes, 
               const magma_int_t nelem, magma_int_t *indices, 
	       const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    magma_readDofs_kernel<<<grid, threads, 0, NULL>>>(NCOMP, nnodes, nelem, 
                                                      indices, du, dv);
}

// ReadDofsTranspose to device memory in tensor dv of size nnodes x NCOMP x nelem
// dv(i, c, e) = du( c, ind(i,e)) 
extern "C" void
magma_readDofsTranspose(const magma_int_t NCOMP,
                        const magma_int_t nnodes,
                        const magma_int_t nelem, magma_int_t *indices,
                        const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    magma_readDofsTranspose_kernel<<<grid, threads, 0, NULL>>>(NCOMP, nnodes, nelem,
                                                               indices, du, dv);
}

// WriteDofs 
// dv( ind(i, e), c) = du(i, c, e)
extern "C" void
magma_writeDofs(const magma_int_t NCOMP, 
                const magma_int_t nnodes, 
                const magma_int_t nelem, magma_int_t *indices, 
	        const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    magma_writeDofs_kernel<<<grid, threads, 0, NULL>>>(NCOMP, nnodes, nelem, 
                                                       indices, du, dv);
}

// WriteDofsTranspose
// dv( c, ind(i,e)) = du(i, c, e)
extern "C" void
magma_writeDofsTranspose(const magma_int_t NCOMP,
                         const magma_int_t nnodes,
                         const magma_int_t nelem, magma_int_t *indices,
                         const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    magma_writeDofsTranspose_kernel<<<grid, threads, 0, NULL>>>(NCOMP, nnodes, nelem,
                                                                indices, du, dv);
}
