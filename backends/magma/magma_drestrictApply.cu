#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory. LLNL-CODE-734707.
// All Rights reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

#include <ceed.h>
#include <magma.h>
#include "atomics.cuh"

//////////////////////////////////////////////////////////////////////////////////////////
// Fastest index listed first
// i : related to nodes
// e : elements
// c: component
// Go from L-vector (du) to E-vector (dv):
//
// dv(i, e, c) = du( ind(i, e), c)  
//         or
// dv(i, e, c) = du(i, e, c)
static __global__ void 
magma_readDofs_kernel(const int NCOMP, const int nnodes, const int esize, const int nelem,
                      int *indices, 
                      const double *du, double *dv)
{
  const int  pid = threadIdx.x;
  const int elem = blockIdx.x;
 
  for (CeedInt i = pid; i < esize; i += blockDim.x) {
        const CeedInt ind = indices ? indices[i + elem * esize] : i + elem * esize;
        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            dv[i+elem*esize+comp*esize*nelem] = du[ind + nnodes * comp];
        }
  }
}

// Fastest index listed first
// i : related to nodes
// e : elements
// c: component
// Go from L-vector (du) to E-vector (dv), with L-vector in transpose format:
//
// dv(i, e, c) = du(c, ind(i, e))  
//         or
// dv(i, e, c) = du(c, i, e)
template<int TBLOCK, int MAXCOMP>
static __global__ void
magma_readDofsTranspose_kernel(const int NCOMP, const int nnodes, const int esize, const int nelem,
                               int *indices,
                               const double *du, double *dv)
{
    const int  pid = threadIdx.x;
    const int elem = blockIdx.x;

    for (CeedInt i = pid; i < esize; i += blockDim.x) {
        const CeedInt ind = indices ? indices[i + elem * esize] : i + elem * esize;
        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            dv[i+elem*esize+comp*esize*nelem] = du[comp + ind * NCOMP];
        }
   }
}

// Fastest index listed first
// i : related to nodes
// e : elements
// c: component
// Go from E-vector (du) to L-vector (dv):
//
// dv(ind(i, e), c) = du(i, e, c)
//         or
// dv(i, e, c) = du(i, e, c)
static __global__ void 
magma_writeDofs_kernel(const int NCOMP, const int nnodes, const int esize, const int nelem,
                      int *indices, 
                      const double *du, double *dv)
{
    const int  pid = threadIdx.x;
    const int elem = blockIdx.x;

    for (CeedInt i = pid; i < esize; i += blockDim.x) {
        const CeedInt ind = indices ? indices[i + elem * esize] : i + elem * esize;
        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            // magmablas_datomic_add(&dv[ind + esize * comp], 
            //                       du[i+comp*esize+elem*NCOMP*esize]);
            magmablas_datomic_add(dv + (ind + nnodes * comp),
                                  du[i+elem*esize+comp*esize*nelem]);
        }
    }
}

// Fastest index listed first
// i : related to nodes
// e : elements
// c: component
// Go from E-vector (du) to L-vector (dv), with L-vector in transpose format:
//
// dv(c, ind(i, e)) = du(i, e, c)
//         or
// dv(c, i, e) = du(i, e, c)
template<int TBLOCK, int MAXCOMP>
static __global__ void
magma_writeDofsTranspose_kernel(const int NCOMP, const int nnodes, const int esize, const int nelem,
                               int *indices,
                               const double *du, double *dv)
{
    const int  pid = threadIdx.x;
    const int elem = blockIdx.x;

    for (CeedInt i = pid; i < esize; i += blockDim.x) {
        const CeedInt ind = indices ? indices[i + elem * esize] : i + elem * esize;
        for (CeedInt comp = 0; comp < NCOMP; ++comp) {
            magmablas_datomic_add(dv + (comp + ind * NCOMP),
                                  du[i+elem*esize+comp*esize*nelem]);
        }
    }
}


//////////////////////////////////////////////////////////////////////////////////////////

// ReadDofs to device memory
// du is L-vector, size nnodes * NCOMP
// dv is E-vector, size nelem * esize * NCOMP
extern "C" void
magma_readDofs(const magma_int_t NCOMP, 
               const magma_int_t nnodes,
               const magma_int_t esize, 
               const magma_int_t nelem, magma_int_t *indices, 
	       const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    magma_readDofs_kernel<<<grid, threads, 0, NULL>>>(NCOMP, nnodes, esize, nelem, 
                                                      indices, du, dv);
}

// ReadDofsTranspose to device memory
// du is L-vector (in tranpose format), size nnodes * NCOMP
// dv is E-vector, size nelem * esize * NCOMP
extern "C" void
magma_readDofsTranspose(const magma_int_t NCOMP,
                        const magma_int_t nnodes,
                        const magma_int_t esize, 
                        const magma_int_t nelem, magma_int_t *indices,
                        const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    assert(NCOMP<=4);
    magma_readDofsTranspose_kernel<256,4><<<grid, threads, 0, NULL>>>(NCOMP, nnodes, esize, nelem,
                                                               indices, du, dv);
}

// WriteDofs from device memory
// du is E-vector, size nelem * esize * NCOMP
// dv is L-vector, size nnodes * NCOMP 
extern "C" void
magma_writeDofs(const magma_int_t NCOMP, 
                const magma_int_t nnodes, 
                const magma_int_t esize, 
                const magma_int_t nelem, magma_int_t *indices, 
	        const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    magma_writeDofs_kernel<<<grid, threads, 0, NULL>>>(NCOMP, nnodes, esize, nelem, 
                                                       indices, du, dv);
}

// WriteDofsTranspose from device memory
// du is E-vector (in transpose format), size nelem * esize * NCOMP
// dv is L-vector, size nnodes * NCOMP 
extern "C" void
magma_writeDofsTranspose(const magma_int_t NCOMP,
                         const magma_int_t nnodes,
                         const magma_int_t esize, 
                         const magma_int_t nelem, magma_int_t *indices,
                         const double *du, double *dv)
{
    magma_int_t grid    = nelem;
    magma_int_t threads = 256;

    assert(NCOMP<=4);
    magma_writeDofsTranspose_kernel<256,4><<<grid, threads, 0, NULL>>>(NCOMP, nnodes, esize, nelem,
                                                                       indices, du, dv);
}
