#include "hip/hip_runtime.h"
// Copyright (c) 2017-2022, Lawrence Livermore National Security, LLC and other CEED contributors.
// All Rights Reserved. See the top-level LICENSE and NOTICE files for details.
//
// SPDX-License-Identifier: BSD-2-Clause
//
// This file is part of CEED:  http://github.com/ceed

#include <ceed.h>
#include <hip/hip_runtime.h>

//------------------------------------------------------------------------------
// Kernel for set value on device
//------------------------------------------------------------------------------
__global__ static void setValueK(CeedScalar * __restrict__ vec, CeedInt size,
                                 CeedScalar val) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  vec[idx] = val;
}

//------------------------------------------------------------------------------
// Set value on device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceSetValue_Cuda(CeedScalar* d_array, CeedInt length,
                                       CeedScalar val) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  setValueK<<<gridsize,bsize>>>(d_array, length, val);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for taking reciprocal
//------------------------------------------------------------------------------
__global__ static void rcpValueK(CeedScalar * __restrict__ vec, CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  if (fabs(vec[idx]) > 1E-16)
    vec[idx] = 1./vec[idx];
}

//------------------------------------------------------------------------------
// Take vector reciprocal in device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceReciprocal_Cuda(CeedScalar* d_array, CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  rcpValueK<<<gridsize,bsize>>>(d_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for scale
//------------------------------------------------------------------------------
__global__ static void scaleValueK(CeedScalar * __restrict__ x, CeedScalar alpha,
    CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  x[idx] *= alpha;
}

//------------------------------------------------------------------------------
// Compute x = alpha x on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceScale_Cuda(CeedScalar *x_array, CeedScalar alpha,
    CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  scaleValueK<<<gridsize,bsize>>>(x_array, alpha, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for axpy
//------------------------------------------------------------------------------
__global__ static void axpyValueK(CeedScalar * __restrict__ y, CeedScalar alpha,
    CeedScalar * __restrict__ x, CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  y[idx] += alpha * x[idx];
}

//------------------------------------------------------------------------------
// Compute y = alpha x + y on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceAXPY_Cuda(CeedScalar *y_array, CeedScalar alpha,
    CeedScalar *x_array, CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  axpyValueK<<<gridsize,bsize>>>(y_array, alpha, x_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for axpby
//------------------------------------------------------------------------------
__global__ static void axpbyValueK(CeedScalar * __restrict__ y, CeedScalar alpha, CeedScalar beta,
    CeedScalar * __restrict__ x, CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  y[idx] = beta * y[idx];
  y[idx] += alpha * x[idx];
}

//------------------------------------------------------------------------------
// Compute y = alpha x + beta y on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceAXPBY_Cuda(CeedScalar *y_array, CeedScalar alpha, CeedScalar beta,
    CeedScalar *x_array, CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  axpbyValueK<<<gridsize,bsize>>>(y_array, alpha, beta, x_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for pointwise mult
//------------------------------------------------------------------------------
__global__ static void pointwiseMultValueK(CeedScalar * __restrict__ w,
    CeedScalar * x, CeedScalar * __restrict__ y, CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  w[idx] = x[idx] * y[idx];
}

//------------------------------------------------------------------------------
// Compute the pointwise multiplication w = x .* y on device
//------------------------------------------------------------------------------
extern "C" int CeedDevicePointwiseMult_Cuda(CeedScalar *w_array, CeedScalar *x_array,
    CeedScalar *y_array, CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  pointwiseMultValueK<<<gridsize,bsize>>>(w_array, x_array, y_array, length);
  return 0;
}
