#include "hip/hip_runtime.h"
// Copyright (c) 2017-2025, Lawrence Livermore National Security, LLC and other CEED contributors.
// All Rights Reserved. See the top-level LICENSE and NOTICE files for details.
//
// SPDX-License-Identifier: BSD-2-Clause
//
// This file is part of CEED:  http://github.com/ceed

#include <ceed.h>
#include <hip/hip_runtime.h>

//------------------------------------------------------------------------------
// Kernel for copy strided on device
//------------------------------------------------------------------------------
__global__ static void copyStridedK(CeedScalar *__restrict__ vec, CeedSize start, CeedSize stop, CeedSize step, CeedScalar *__restrict__ vec_copy) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < stop - start) {
    if (index % step == 0) vec_copy[start + index] = vec[start + index];
  }
}

//------------------------------------------------------------------------------
// Copy strided on device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceCopyStrided_Cuda(CeedScalar *d_array, CeedSize start, CeedSize stop, CeedSize step, CeedScalar *d_copy_array) {
  const int      block_size = 512;
  const CeedSize copy_size  = stop - start;
  int            grid_size  = copy_size / block_size;

  if (block_size * grid_size < copy_size) grid_size += 1;
  copyStridedK<<<grid_size, block_size>>>(d_array, start, stop, step, d_copy_array);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for set value on device
//------------------------------------------------------------------------------
__global__ static void setValueK(CeedScalar *__restrict__ vec, CeedSize size, CeedScalar val) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < size) vec[index] = val;
}

//------------------------------------------------------------------------------
// Set value on device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceSetValue_Cuda(CeedScalar *d_array, CeedSize length, CeedScalar val) {
  const int      block_size = 512;
  const CeedSize vec_size   = length;
  int            grid_size  = vec_size / block_size;

  if (block_size * grid_size < vec_size) grid_size += 1;
  setValueK<<<grid_size, block_size>>>(d_array, length, val);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for set value strided on device
//------------------------------------------------------------------------------
__global__ static void setValueStridedK(CeedScalar *__restrict__ vec, CeedSize start, CeedSize stop, CeedSize step, CeedScalar val) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < stop - start) {
    if (index % step == 0) vec[start + index] = val;
  }
}

//------------------------------------------------------------------------------
// Set value strided on device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceSetValueStrided_Cuda(CeedScalar *d_array, CeedSize start, CeedSize stop, CeedSize step, CeedScalar val) {
  const int      block_size = 512;
  const CeedSize set_size   = stop - start;
  int            grid_size  = set_size / block_size;

  if (block_size * grid_size < set_size) grid_size += 1;
  setValueStridedK<<<grid_size, block_size>>>(d_array, start, stop, step, val);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for taking reciprocal
//------------------------------------------------------------------------------
__global__ static void rcpValueK(CeedScalar *__restrict__ vec, CeedSize size) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < size) {
    if (fabs(vec[index]) > 1E-16) vec[index] = 1. / vec[index];
  }
}

//------------------------------------------------------------------------------
// Take vector reciprocal in device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceReciprocal_Cuda(CeedScalar *d_array, CeedSize length) {
  const int      block_size = 512;
  const CeedSize vec_size   = length;
  int            grid_size  = vec_size / block_size;

  if (block_size * grid_size < vec_size) grid_size += 1;
  rcpValueK<<<grid_size, block_size>>>(d_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for scale
//------------------------------------------------------------------------------
__global__ static void scaleValueK(CeedScalar *__restrict__ x, CeedScalar alpha, CeedSize size) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < size) x[index] *= alpha;
}

//------------------------------------------------------------------------------
// Compute x = alpha x on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceScale_Cuda(CeedScalar *x_array, CeedScalar alpha, CeedSize length) {
  const int      block_size = 512;
  const CeedSize vec_size   = length;
  int            grid_size  = vec_size / block_size;

  if (block_size * grid_size < vec_size) grid_size += 1;
  scaleValueK<<<grid_size, block_size>>>(x_array, alpha, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for axpy
//------------------------------------------------------------------------------
__global__ static void axpyValueK(CeedScalar *__restrict__ y, CeedScalar alpha, CeedScalar *__restrict__ x, CeedSize size) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < size) y[index] += alpha * x[index];
}

//------------------------------------------------------------------------------
// Compute y = alpha x + y on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceAXPY_Cuda(CeedScalar *y_array, CeedScalar alpha, CeedScalar *x_array, CeedSize length) {
  const int      block_size = 512;
  const CeedSize vec_size   = length;
  int            grid_size  = vec_size / block_size;

  if (block_size * grid_size < vec_size) grid_size += 1;
  axpyValueK<<<grid_size, block_size>>>(y_array, alpha, x_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for axpby
//------------------------------------------------------------------------------
__global__ static void axpbyValueK(CeedScalar *__restrict__ y, CeedScalar alpha, CeedScalar beta, CeedScalar *__restrict__ x, CeedSize size) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < size) {
    y[index] = beta * y[index];
    y[index] += alpha * x[index];
  }
}

//------------------------------------------------------------------------------
// Compute y = alpha x + beta y on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceAXPBY_Cuda(CeedScalar *y_array, CeedScalar alpha, CeedScalar beta, CeedScalar *x_array, CeedSize length) {
  const int      block_size = 512;
  const CeedSize vec_size   = length;
  int            grid_size  = vec_size / block_size;

  if (block_size * grid_size < vec_size) grid_size += 1;
  axpbyValueK<<<grid_size, block_size>>>(y_array, alpha, beta, x_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for pointwise mult
//------------------------------------------------------------------------------
__global__ static void pointwiseMultValueK(CeedScalar *__restrict__ w, CeedScalar *x, CeedScalar *__restrict__ y, CeedSize size) {
  const CeedSize index = threadIdx.x + (CeedSize)blockDim.x * blockIdx.x;

  if (index < size) w[index] = x[index] * y[index];
}

//------------------------------------------------------------------------------
// Compute the pointwise multiplication w = x .* y on device
//------------------------------------------------------------------------------
extern "C" int CeedDevicePointwiseMult_Cuda(CeedScalar *w_array, CeedScalar *x_array, CeedScalar *y_array, CeedSize length) {
  const int      block_size = 512;
  const CeedSize vec_size   = length;
  int            grid_size  = vec_size / block_size;

  if (block_size * grid_size < vec_size) grid_size += 1;
  pointwiseMultValueK<<<grid_size, block_size>>>(w_array, x_array, y_array, length);
  return 0;
}

//------------------------------------------------------------------------------
