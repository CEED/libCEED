// Copyright (c) 2017, Lawrence Livermore National Security, LLC. Produced at
// the Lawrence Livermore National Laboratory. LLNL-CODE-734707. All Rights
// reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.
#include "../include/ceed.h"
#include <hip/hip_runtime.h>

const int sizeMax = 16;
__constant__ double c_B[sizeMax*sizeMax];
__constant__ double c_G[sizeMax*sizeMax];

extern "C" int CeedCudaInitInterp(CeedScalar *d_B, CeedInt P1d, CeedInt Q1d,
                                  CeedScalar **c_B_ptr) {
  const int Bsize = P1d*Q1d*sizeof(CeedScalar);
  hipMemcpyToSymbol(HIP_SYMBOL(c_B), d_B, Bsize, 0, hipMemcpyDeviceToDevice);
  hipGetSymbolAddress((void **)c_B_ptr, c_B);
  return 0;
}

extern "C" int CeedCudaInitInterpGrad(CeedScalar *d_B, CeedScalar *d_G,
    CeedInt P1d, CeedInt Q1d, CeedScalar **c_B_ptr, CeedScalar **c_G_ptr) {
  const int Bsize = P1d*Q1d*sizeof(CeedScalar);
  hipMemcpyToSymbol(HIP_SYMBOL(c_B), d_B, Bsize, 0, hipMemcpyDeviceToDevice);
  hipGetSymbolAddress((void **)c_B_ptr, c_B);
  hipMemcpyToSymbol(HIP_SYMBOL(c_G), d_G, Bsize, 0, hipMemcpyDeviceToDevice);
  hipGetSymbolAddress((void **)c_G_ptr, c_G);
  return 0;
}
