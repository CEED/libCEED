#include "hip/hip_runtime.h"
// Copyright (c) 2017, Lawrence Livermore National Security, LLC. Produced at
// the Lawrence Livermore National Laboratory. LLNL-CODE-734707. All Rights
// reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

#include <ceed/ceed.h>
#include <hip/hip_runtime.h>

//------------------------------------------------------------------------------
// Kernel for set value on device
//------------------------------------------------------------------------------
__global__ static void setValueK(CeedScalar * __restrict__ vec, CeedInt size,
                                 CeedScalar val) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  vec[idx] = val;
}

//------------------------------------------------------------------------------
// Set value on device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceSetValue_Cuda(CeedScalar* d_array, CeedInt length,
                                       CeedScalar val) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  setValueK<<<gridsize,bsize>>>(d_array, length, val);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for taking reciprocal
//------------------------------------------------------------------------------
__global__ static void rcpValueK(CeedScalar * __restrict__ vec, CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  if (fabs(vec[idx]) > 1E-16)
    vec[idx] = 1./vec[idx];
}

//------------------------------------------------------------------------------
// Take vector reciprocal in device memory
//------------------------------------------------------------------------------
extern "C" int CeedDeviceReciprocal_Cuda(CeedScalar* d_array, CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  rcpValueK<<<gridsize,bsize>>>(d_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for scale
//------------------------------------------------------------------------------
__global__ static void scaleValueK(CeedScalar * __restrict__ x, CeedScalar alpha,
    CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  x[idx] *= alpha;
}

//------------------------------------------------------------------------------
// Compute x = alpha x on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceScale_Cuda(CeedScalar *x_array, CeedScalar alpha,
    CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  scaleValueK<<<gridsize,bsize>>>(x_array, alpha, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for axpy
//------------------------------------------------------------------------------
__global__ static void axpyValueK(CeedScalar * __restrict__ y, CeedScalar alpha,
    CeedScalar * __restrict__ x, CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  y[idx] += alpha * x[idx];
}

//------------------------------------------------------------------------------
// Compute y = alpha x + y on device
//------------------------------------------------------------------------------
extern "C" int CeedDeviceAXPY_Cuda(CeedScalar *y_array, CeedScalar alpha,
    CeedScalar *x_array, CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  axpyValueK<<<gridsize,bsize>>>(y_array, alpha, x_array, length);
  return 0;
}

//------------------------------------------------------------------------------
// Kernel for pointwise mult
//------------------------------------------------------------------------------
__global__ static void pointwiseMultValueK(CeedScalar * __restrict__ w,
    CeedScalar * x, CeedScalar * __restrict__ y, CeedInt size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= size)
    return;
  w[idx] = x[idx] * y[idx];
}

//------------------------------------------------------------------------------
// Compute the pointwise multiplication w = x .* y on device
//------------------------------------------------------------------------------
extern "C" int CeedDevicePointwiseMult_Cuda(CeedScalar *w_array, CeedScalar *x_array,
    CeedScalar *y_array, CeedInt length) {
  const int bsize = 512;
  const int vecsize = length;
  int gridsize = vecsize / bsize;

  if (bsize * gridsize < vecsize)
    gridsize += 1;
  pointwiseMultValueK<<<gridsize,bsize>>>(w_array, x_array, y_array, length);
  return 0;
}
