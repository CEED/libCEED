#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory. LLNL-CODE-734707.
// All Rights reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

#include <ceed-impl.h>
#include <string.h>
#include "ceed-cuda.cuh"

static __global__ void noTrScalar(const CeedInt esize, const CeedInt * __restrict__ indices, const CeedScalar * __restrict__ u, CeedScalar * __restrict__ v) {
  for (CeedInt i = blockIdx.x*blockDim.x + threadIdx.x; i < esize; i += blockDim.x * gridDim.x) {
    v[i] = u[indices[i]];
  }
}

static __global__ void noTrNoTr(const CeedInt esize, const CeedInt ncomp, const CeedInt elemsize, const CeedInt nelem, const CeedInt ndof, const CeedInt * __restrict__ indices, const CeedScalar * __restrict__ u, CeedScalar * __restrict__ v) {
  for (CeedInt i = blockIdx.x*blockDim.x + threadIdx.x; i < esize; i += blockDim.x * gridDim.x) {
    const CeedInt e = i / (ncomp * elemsize);
    const CeedInt d = (i / elemsize) % ncomp;
    const CeedInt s = i % elemsize;

    v[i] = u[indices[s + elemsize * e] + ndof * d];
  }
}

static __global__ void noTrTr(const CeedInt esize, const CeedInt ncomp, const CeedInt elemsize, const CeedInt nelem, const CeedInt * __restrict__ indices, const CeedScalar * __restrict__ u, CeedScalar * __restrict__ v) {
  for (CeedInt i = blockIdx.x*blockDim.x + threadIdx.x; i < esize; i += blockDim.x * gridDim.x) {
    const CeedInt e = i / (ncomp * elemsize);
    const CeedInt d = (i / elemsize) % ncomp;
    const CeedInt s = i % elemsize;

    v[i] = u[ncomp * indices[s + elemsize * e] + d];
  }
}

static __global__ void trScalar(const CeedInt esize, const CeedInt * __restrict__ indices, const CeedScalar * __restrict__ u, CeedScalar * __restrict__ v) {
  for (CeedInt i = blockIdx.x*blockDim.x + threadIdx.x; i < esize; i += blockDim.x * gridDim.x) {
    atomicAdd(v + indices[i], u[i]);
  }
}

static __global__ void trNoTr(const CeedInt esize, const CeedInt ncomp, const CeedInt elemsize, const CeedInt nelem, const CeedInt ndof, const CeedInt * __restrict__ indices, const CeedScalar * __restrict__ u, CeedScalar * __restrict__ v) {
  for (CeedInt i = blockIdx.x*blockDim.x + threadIdx.x; i < esize; i += blockDim.x * gridDim.x) {
    const CeedInt e = i / (ncomp * elemsize);
    const CeedInt d = (i / elemsize) % ncomp;
    const CeedInt s = i % elemsize;

    atomicAdd(v + (indices[s + elemsize * e] + ndof * d), u[i]);
  }
}

static __global__ void trTr(const CeedInt ncomp, const CeedInt esize, const CeedInt elemsize, const CeedInt nelem, const CeedInt * __restrict__ indices, const CeedScalar * __restrict__ u, CeedScalar * __restrict__ v) {
  for (CeedInt i = blockIdx.x*blockDim.x + threadIdx.x; i < esize; i += blockDim.x * gridDim.x) {
    const CeedInt e = i / (ncomp * elemsize);
    const CeedInt d = (i / elemsize) % ncomp;
    const CeedInt s = i % elemsize;

    atomicAdd(v + (ncomp * indices[s + elemsize * e] + d), u[i]);
  }
}

static int CeedElemRestrictionApply_Cuda(CeedElemRestriction r,
    CeedTransposeMode tmode, CeedTransposeMode lmode,
    CeedVector u, CeedVector v, CeedRequest *request) {
  CeedElemRestriction_Cuda *impl = (CeedElemRestriction_Cuda*)r->data;
  Ceed_Cuda *data = (Ceed_Cuda*)r->ceed->data;
  int ierr;
  const CeedInt nelem = r->nelem;
  const CeedInt ncomp = r->ncomp;
  const CeedInt elemsize = r->elemsize;
  const CeedInt ndof = r->ndof;
  const CeedInt esize = nelem*elemsize*ncomp;
  const CeedScalar *d_u;
  CeedScalar *d_v;
  ierr = CeedVectorGetArrayRead(u, CEED_MEM_DEVICE, &d_u); CeedChk(ierr);
  ierr = CeedVectorGetArray(v, CEED_MEM_DEVICE, &d_v); CeedChk(ierr);

  if (!impl->indices) {
    hipMemcpy(d_v, d_u, esize * sizeof(CeedScalar), hipMemcpyDeviceToDevice);
  } else {
    const CeedInt* d_indices;
    ierr = CeedVectorGetArrayRead(impl->indices, CEED_MEM_DEVICE, (const CeedScalar**)&d_indices); CeedChk(ierr);
    if (tmode == CEED_NOTRANSPOSE) {
      // Perform: v = r * u
      if (ncomp == 1) {
        ierr = run_cuda(noTrScalar, data->optBlockSize, 0, esize, d_indices, d_u, d_v); CeedChk(ierr);
        // vv is (elemsize x ncomp x nelem), column-major
      } else if (lmode == CEED_NOTRANSPOSE) {
        // u is (ndof x ncomp), column-major
        ierr = run_cuda(noTrNoTr, data->optBlockSize, 0, esize, ncomp, elemsize, nelem, ndof, d_indices, d_u, d_v); CeedChk(ierr);
      } else { // u is (ncomp x ndof), column-major
        ierr = run_cuda(noTrTr, data->optBlockSize, 0, esize, ncomp, elemsize, nelem, d_indices, d_u, d_v); CeedChk(ierr);
      }
    } else {
      // Note: in transpose mode, we perform: v += r^t * u
      if (ncomp == 1) {
        ierr = run_cuda(trScalar, data->optBlockSize, 0, esize, d_indices, d_u, d_v); CeedChk(ierr);
        // u is (elemsize x ncomp x nelem)
      } else if (lmode == CEED_NOTRANSPOSE) {
        // vv is (ndof x ncomp), column-major
        ierr = run_cuda(trNoTr, data->optBlockSize, 0, esize, ncomp, elemsize, nelem, ndof, d_indices, d_u, d_v); CeedChk(ierr);
      } else { // vv is (ncomp x ndof), column-major
        ierr = run_cuda(trTr, data->optBlockSize, 0, esize, ncomp, elemsize, nelem, d_indices, d_u, d_v); CeedChk(ierr);
      }
    }
  } 
  if (request != CEED_REQUEST_IMMEDIATE && request != CEED_REQUEST_ORDERED)
    *request = NULL;

  return 0;
}

static int CeedElemRestrictionDestroy_Cuda(CeedElemRestriction r) {
  CeedElemRestriction_Cuda *impl = (CeedElemRestriction_Cuda*)r->data;
  int ierr;

  ierr = CeedVectorDestroy(&impl->indices); CeedChk(ierr);
  ierr = CeedFree(&r->data); CeedChk(ierr);
  return 0;
}

int CeedElemRestrictionCreate_Cuda(CeedElemRestriction r,
    CeedMemType mtype,
    CeedCopyMode cmode, const CeedInt *indices) {
  int ierr;
  CeedElemRestriction_Cuda *impl;
  ierr = CeedCalloc(1,&impl); CeedChk(ierr);
  if (indices) {
    ierr = CeedVectorCreate(r->ceed, r->nelem*r->elemsize*sizeof(CeedInt)/sizeof(CeedScalar) + 1, &impl->indices); CeedChk(ierr);
    ierr = CeedVectorSetArray(impl->indices, mtype, cmode, (CeedScalar*)indices); CeedChk(ierr);
  } else {
    impl->indices = NULL;
  }

  r->data = impl;
  r->Apply = CeedElemRestrictionApply_Cuda;
  r->Destroy = CeedElemRestrictionDestroy_Cuda;
  return 0;
}
