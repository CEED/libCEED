#include "hip/hip_runtime.h"
// Copyright (c) 2017-2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory. LLNL-CODE-734707.
// All Rights reserved. See files LICENSE and NOTICE for details.
//
// This file is part of CEED, a collection of benchmarks, miniapps, software
// libraries and APIs for efficient high-order finite element and spectral
// element discretizations for exascale applications. For more information and
// source code availability see http://github.com/ceed.
//
// The CEED research is supported by the Exascale Computing Project 17-SC-20-SC,
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.

#include <ceed-impl.h>
#include <string.h>
#include "ceed-cuda.cuh"

static inline size_t bytes(const CeedElemRestriction res) {
  return res->nelem * res->elemsize * sizeof(CeedInt);
}

__global__ void noTrScalar(const CeedInt esize, const CeedInt *indices, const CeedScalar* u, CeedScalar* v) {
  const CeedInt i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < esize) {
    v[i] = u[indices[i]];
  }
}

__global__ void noTrNoTr(const CeedInt nelem, const CeedInt ncomp, const CeedInt elemsize, const CeedInt ndof, const CeedInt *indices, const CeedScalar* u, CeedScalar* v) {
  const CeedInt i = blockIdx.z*blockDim.z + threadIdx.z;
  const CeedInt d = blockIdx.y*blockDim.y + threadIdx.y;
  const CeedInt e = blockIdx.x*blockDim.x + threadIdx.x;

  if (e >= nelem || d >= ncomp || i >= elemsize) {
    return;
  }
  
  v[i + elemsize * (d + ncomp * e)] = u[indices[i + elemsize * e] + ndof * d];
}

__global__ void noTrTr(const CeedInt nelem, const CeedInt ncomp, const CeedInt elemsize, const CeedInt *indices, const CeedScalar* u, CeedScalar* v) {
  const CeedInt i = blockIdx.z*blockDim.z + threadIdx.z;
  const CeedInt d = blockIdx.y*blockDim.y + threadIdx.y;
  const CeedInt e = blockIdx.x*blockDim.x + threadIdx.x;

  if (e >= nelem || d >= ncomp || i >= elemsize) {
    return;
  }
  
  v[i + elemsize * (d + ncomp * e)] = u[d + ncomp * indices[i + elemsize * e]];
}

__global__ void trScalar(const CeedInt esize, const CeedInt *indices, const CeedScalar* u, CeedScalar* v) {
  const CeedInt i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < esize) {
    atomicAdd(v + indices[i], u[i]);
  }
}

__global__ void trNoTr(const CeedInt nelem, const CeedInt ncomp, const CeedInt elemsize, const CeedInt ndof, const CeedInt *indices, const CeedScalar* u, CeedScalar* v) {
  const CeedInt i = blockIdx.z*blockDim.z + threadIdx.z;
  const CeedInt d = blockIdx.y*blockDim.y + threadIdx.y;
  const CeedInt e = blockIdx.x*blockDim.x + threadIdx.x;

  if (e >= nelem || d >= ncomp || i >= elemsize) {
    return;
  }

  atomicAdd(v + (indices[i+elemsize*e]+ndof*d), u[i+elemsize*(d+e*ncomp)]);
}

__global__ void trTr(const CeedInt nelem, const CeedInt ncomp, const CeedInt elemsize, const CeedInt *indices, const CeedScalar* u, CeedScalar* v) {
  const CeedInt i = blockIdx.z*blockDim.z + threadIdx.z;
  const CeedInt d = blockIdx.y*blockDim.y + threadIdx.y;
  const CeedInt e = blockIdx.x*blockDim.x + threadIdx.x;

  if (e >= nelem || d >= ncomp || i >= elemsize) {
    return;
  }
  atomicAdd(v + (d+ncomp*indices[i+elemsize*e]), u[i+elemsize*(d+e*ncomp)]);
}



static int CeedElemRestrictionApply_Cuda(CeedElemRestriction r,
                                        CeedTransposeMode tmode, CeedInt ncomp,
                                        CeedTransposeMode lmode, CeedVector u,
                                        CeedVector v, CeedRequest *request) {
  CeedElemRestriction_Cuda *impl = (CeedElemRestriction_Cuda*)r->data;
  Ceed_Cuda *data = (Ceed_Cuda*)r->ceed->data;
  int ierr;
  const CeedInt nelem = r->nelem;
  const CeedInt elemsize = r->elemsize;
  const CeedInt ndof = r->ndof;
  const CeedInt esize = nelem*elemsize;
  const CeedInt *d_indices = impl->d_indices;
  const CeedScalar *d_u = ((CeedVector_Cuda*)u->data)->d_array;
  CeedScalar *d_v = ((CeedVector_Cuda*)v->data)->d_array;

  ierr = CeedVectorGetArrayRead(u, CEED_MEM_HOST, &d_u); CeedChk(ierr);
  ierr = CeedVectorGetArray(v, CEED_MEM_HOST, &d_v); CeedChk(ierr);
  if (tmode == CEED_NOTRANSPOSE) {
    // Perform: v = r * u
    if (ncomp == 1) {
      run1d(data, noTrScalar, esize, d_indices, d_u, d_v);
    } else {
      // vv is (elemsize x ncomp x nelem), column-major
      if (lmode == CEED_NOTRANSPOSE) { // u is (ndof x ncomp), column-major
        run3d(data, noTrNoTr, nelem, ncomp, elemsize, ndof, d_indices, d_u, d_v);
      } else { // u is (ncomp x ndof), column-major
        run3d(data, noTrTr, nelem, ncomp, elemsize, d_indices, d_u, d_v);
      }
    }
  } else {
    // Note: in transpose mode, we perform: v += r^t * u
    if (ncomp == 1) {
      run1d(data, trScalar, elemsize, d_indices, d_u, d_v);
    } else {
      // u is (elemsize x ncomp x nelem)
      if (lmode == CEED_NOTRANSPOSE) { // vv is (ndof x ncomp), column-major
        run3d(data, trNoTr, nelem, ncomp, elemsize, ndof, d_indices, d_u, d_v);
      } else { // vv is (ncomp x ndof), column-major
        run3d(data, trTr, nelem, ncomp, elemsize, d_indices, d_u, d_v);
      }
    }
  }
  CeedChk(hipGetLastError());
  if (request != CEED_REQUEST_IMMEDIATE && request != CEED_REQUEST_ORDERED)
    *request = NULL;
  return 0;
}

static int CeedElemRestrictionDestroy_Cuda(CeedElemRestriction r) {
  CeedElemRestriction_Cuda *impl = (CeedElemRestriction_Cuda*)r->data;
  int ierr;

  ierr = hipFree(impl->d_indices); CeedChk(ierr);
  ierr = CeedFree(&r->data); CeedChk(ierr);
  return 0;
}

int CeedElemRestrictionCreate_Cuda(CeedElemRestriction r,
                                  CeedMemType mtype,
                                  CeedCopyMode cmode, const CeedInt *indices) {
  int ierr;
  CeedElemRestriction_Cuda *impl;

  if (mtype != CEED_MEM_HOST)
    return CeedError(r->ceed, 1, "Only MemType = HOST supported");
  ierr = CeedCalloc(1,&impl); CeedChk(ierr);

  ierr = hipMalloc(&impl->d_indices, bytes(r)); CeedChk(ierr);
  ierr = hipMemcpy(impl->d_indices, indices, bytes(r), hipMemcpyHostToDevice); CeedChk(ierr);

  r->data = impl;
  r->Apply = CeedElemRestrictionApply_Cuda;
  r->Destroy = CeedElemRestrictionDestroy_Cuda;
  return 0;
}
